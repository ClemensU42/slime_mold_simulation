#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <cstdint>

#include "lodepng.h"
#include <raylib.h>

#define SENSOR_DIST_FROM_AGENT 5
#define SENSOR_SIZE_FROM_AGENT 5
#define DIFFUSE_RADIUS 4
#define EVAPORATION_SPEED 0.75
#define DIFFUSE_SPEED 50
#define TRAIL_STRENGTH 0.125

struct agent{
	float xPos, yPos;
	float rotation;
	float speed;
};

__device__
float lerp(float a, float b, float t){
	return a + t * (b - a);
}

__global__
void updateAgents(int n, agent* agents, float* floatImage, int imageWidth, int imageHeight, float delatTime){
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for(int i = index; i < n; i += stride){
		float sense0sum, sense1sum, sense2sum;
		int sensor0xPos = floor(agents[i].xPos + (sin(agents[i].rotation * DEG2RAD + 45 * DEG2RAD) *
				SENSOR_DIST_FROM_AGENT));
		int sensor0yPos = floor(agents[i].yPos + (cos(agents[i].rotation * DEG2RAD + 45 * DEG2RAD) *
				SENSOR_DIST_FROM_AGENT));
		int sensor1xPos = floor(agents[i].xPos + (sin(agents[i].rotation * DEG2RAD) * SENSOR_DIST_FROM_AGENT));
		int sensor1yPos = floor(agents[i].yPos + (cos(agents[i].rotation * DEG2RAD) * SENSOR_DIST_FROM_AGENT));
		int sensor2xPos = floor(agents[i].xPos + (sin(agents[i].rotation * DEG2RAD - 45 * DEG2RAD) *
				SENSOR_DIST_FROM_AGENT));
		int sensor2yPos = floor(agents[i].yPos + (cos(agents[i].rotation * DEG2RAD - 45 * DEG2RAD) *
				SENSOR_DIST_FROM_AGENT));

		for(int x = -SENSOR_SIZE_FROM_AGENT; x < SENSOR_SIZE_FROM_AGENT; x++){
			for(int y = -SENSOR_SIZE_FROM_AGENT; y < SENSOR_SIZE_FROM_AGENT; y++){
				int s0px = sensor0xPos + x;
				int s0py = sensor0yPos + y;
				int s1px = sensor1xPos + x;
				int s1py = sensor1yPos + y;
				int s2px = sensor2xPos + x;
				int s2py = sensor2yPos + y;

				if(s0px >= 0 && s0px < imageWidth && s0py >= 0 && s0py < imageHeight)
					sense0sum += floatImage[s0px + s0py * imageWidth];
				if(s1px >= 0 && s1px < imageWidth && s1py >= 0 && s1py < imageHeight)
					sense1sum += floatImage[s1px + s1py * imageWidth];
				if(s2px >= 0 && s2px < imageWidth && s2py >= 0 && s2py < imageHeight)
					sense2sum += floatImage[s2px + s2py * imageWidth];

			}
		}

		if(sense0sum > sense1sum && sense0sum > sense2sum)
			agents[i].rotation += 360 * delatTime;
		else if(sense2sum > sense1sum && sense2sum > sense0sum)
			agents[i].rotation -= 360 * delatTime;

		agents[i].xPos += sin(agents[i].rotation * DEG2RAD) * agents[i].speed * delatTime;
		agents[i].yPos += cos(agents[i].rotation * DEG2RAD) * agents[i].speed * delatTime;

		if(agents[i].xPos <= 0 || agents[i].xPos >= imageWidth || agents[i].yPos <= 0 || agents[i].yPos >= imageHeight)
		{
			agents[i].rotation += 180;
			agents[i].xPos = max(0.0f, min(agents[i].xPos, imageWidth - 1.0f));
			agents[i].yPos = max(0.0f, min(agents[i].yPos, imageWidth - 1.0f));
		}
	}
}

__global__
void leaveTrail(int n, agent* agents, float* floatImage, int imageWidth, int imageHeight){
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for(int i = index; i < n; i += stride){
		int x = floor(agents[i].xPos);
		int y = floor(agents[i].yPos);
		floatImage[x + y * imageWidth] = max(0.0f, min(1.0f, floatImage[x + y * imageWidth] + TRAIL_STRENGTH));
	}
}

__global__
void diffuseTrail(float* currentFloatImage, float* newFloatImage, int imageSize, int imageWidth, float deltaTime){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	float sum = 0;
	int hits = 0;

	for(int dx = x - 1; dx <= x + 1; dx++)
		for(int dy = y - 1; dy <= y + 1; dy++)
			if(dx >= 0 && dx < imageWidth && dy >= 0 && dy < imageSize / imageWidth)
				sum += currentFloatImage[dx + dy * imageWidth]; hits++;
	float blurResult = sum / 9.0f;

	float diffuseWeight = saturate(DIFFUSE_SPEED * deltaTime);

	float diffusedValue = currentFloatImage[x + y * imageWidth] * (1 - diffuseWeight) + blurResult * diffuseWeight;

	newFloatImage[x + y * imageWidth] = max(0.0f, diffusedValue - EVAPORATION_SPEED * deltaTime);
}

__global__
void floatImageToRGBAByteImage(float* floatImage, uint8_t* rgbaImage, int imageSize, int imageWidth){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	int index = x * 4 + y * imageWidth * 4;
	int floatIndex = x + y * imageWidth;
	rgbaImage[index] = (floatImage[floatIndex] * 255);
	rgbaImage[index + 1] = (floatImage[floatIndex] * 255);
	rgbaImage[index + 2] = (floatImage[floatIndex] * 255);
	rgbaImage[index + 3] = 255;
}

int main()
{
	std::string filePath = "./frames/";
	int numAgents = 500000;
	float* currentFloatImage;
	float* newFloatImage;
	uint8_t* gpuRGBAImage;
	uint8_t* hostRGBAImage;
	agent* gpuAgents;
	agent* hostAgents;
	int imageWidth = 1920;
	int imageHeight = 1080;
	int frameAmount = 250;

	system("mkdir -p ./frames");

	hostRGBAImage = (uint8_t*)malloc(imageWidth * imageHeight * 4);
	hostAgents = (agent*)malloc(numAgents * sizeof(agent));

	hipMalloc(&gpuRGBAImage, imageWidth * imageHeight * 4);
	hipMalloc(&gpuAgents, numAgents * sizeof(agent));
	hipMalloc(&currentFloatImage, imageWidth * imageHeight * sizeof(float));
	hipMalloc(&newFloatImage, imageWidth * imageHeight * sizeof(float));

	hipMemset(currentFloatImage, 0, imageWidth * imageHeight * sizeof(float));
	hipMemset(newFloatImage, 0, imageWidth * imageHeight * sizeof(float));

	for(int i = 0; i < numAgents; i++){
		hostAgents[i].xPos = imageWidth / 2 + (rand() % 600 - 300);
		hostAgents[i].yPos = imageHeight / 2 + (rand() % 600 - 300);
		//hostAgents[i].xPos = imageWidth / 2;
		//hostAgents[i].yPos = imageHeight / 2;
		hostAgents[i].rotation = atan2(imageWidth / 2 - hostAgents[i].xPos, imageHeight / 2 - hostAgents[i].yPos) - 90;
		hostAgents[i].speed = 50;
	}

	hipMemcpy(gpuAgents, hostAgents, numAgents * sizeof(agent), hipMemcpyHostToDevice);

	/*
	for(int frame = 0; frame < frameAmount; frame++)
	{
		std::cout << "rendering image no:" << frame << std::endl;
		int blockSize = 256;
		int gridSize = ceil(numAgents / blockSize) + 1;

		updateAgents<<<gridSize, blockSize>>>(numAgents, gpuAgents, currentFloatImage, imageWidth, imageHeight);
		hipDeviceSynchronize();

		leaveTrail<<<gridSize, blockSize>>>(numAgents, gpuAgents, currentFloatImage, imageWidth, imageHeight);
		hipDeviceSynchronize();

		dim3 blockDim(32, 32);
		dim3 gridDim(ceil(imageWidth / blockDim.x), ceil(imageHeight / blockDim.y) + 1);

		diffuseTrail<<<gridDim, blockDim>>>(currentFloatImage, newFloatImage, imageWidth * imageHeight, imageWidth);
		hipDeviceSynchronize();

		float* temp = currentFloatImage;
		currentFloatImage = newFloatImage;
		newFloatImage = temp;

		floatImageToRGBAByteImage<<<gridDim, blockDim>>>(currentFloatImage, gpuRGBAImage, imageWidth * imageHeight,
		                                                 imageWidth);
		hipDeviceSynchronize();

		hipMemcpy(hostRGBAImage, gpuRGBAImage, imageWidth * imageHeight * 4, hipMemcpyDeviceToHost);

		//lodepng::encode(filePath + std::to_string(frame) + ".png", hostRGBAImage, imageWidth, imageHeight);
	}
	 */

	InitWindow(1920, 1080, "Slime Mold Simulation");
	Image image = GenImageColor(imageWidth, imageHeight, RAYWHITE);

	Texture2D texture = LoadTextureFromImage(image);

	// raylib
	while(!WindowShouldClose())
	{
		int blockSize = 512;
		int gridSize = (numAgents + blockSize - 1) / blockSize;

		updateAgents<<<gridSize, blockSize>>>(numAgents, gpuAgents, currentFloatImage, imageWidth, imageHeight,
											  GetFrameTime());
		hipDeviceSynchronize();

		leaveTrail<<<gridSize, blockSize>>>(numAgents, gpuAgents, currentFloatImage, imageWidth, imageHeight);
		hipDeviceSynchronize();

		dim3 blockDim(32, 32);
		dim3 gridDim(ceil(imageWidth / blockDim.x), ceil(imageHeight / blockDim.y) + 1);

		diffuseTrail<<<gridDim, blockDim>>>(currentFloatImage, newFloatImage, imageWidth * imageHeight, imageWidth,
											GetFrameTime());
		hipDeviceSynchronize();

		float* temp = currentFloatImage;
		currentFloatImage = newFloatImage;
		newFloatImage = temp;

		floatImageToRGBAByteImage<<<gridDim, blockDim>>>(currentFloatImage, gpuRGBAImage, imageWidth * imageHeight,
		                                                 imageWidth);
		hipDeviceSynchronize();

		hipMemcpy(hostRGBAImage, gpuRGBAImage, imageWidth * imageHeight * 4, hipMemcpyDeviceToHost);

		UpdateTexture(texture, hostRGBAImage);

		BeginDrawing();
		ClearBackground(RAYWHITE);
		DrawTexture(texture, 0, 0, WHITE);
		DrawFPS(100, 100);
		EndDrawing();
	}

	CloseWindow();

	hipFree(gpuAgents);
	hipFree(gpuRGBAImage);
	hipFree(currentFloatImage);
	hipFree(newFloatImage);

	free(hostAgents);
	free(hostRGBAImage);
	return 0;
}
